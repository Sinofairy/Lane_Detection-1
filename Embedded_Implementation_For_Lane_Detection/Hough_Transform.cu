#include "hip/hip_runtime.h"
/*
*Test Code for hough transform using CUDA
*Adapted From "Fast Hough Trasform on GPU's"
*
*
*
*
*/
#include"hough.hpp"
bool debug = false;
#define THREADS_X 	32
#define THREADS_Y	4
#define PIXELS_PER_THREAD 16
#define BLOCKS_X 	(IMG_WIDTH  / (THREADS_X*PIXELS_PER_THREAD))
#define BLOCKS_Y 	(IMG_HEIGHT / THREADS_Y)
#define MAX_QUEUE_LENGTH (THREADS_X*THREADS_Y*PIXELS_PER_THREAD)


void print_array(float *arr, int size)
{
	for(int i =0;i<size;i++)
	{
		cout<<*(arr + i)<<"\t";
	}

	cout<<endl;

}

void print_image(unsigned char *image, int height, int width)
{


	for(int i =0;i<height;i++)
	{
		for(int j =0;j<width;j++)
		{
			cout<<(int)*(image + i*width + j)<<"\t";

		}
	
		cout<<endl;
	}

}


void print_houghspace(unsigned int* const array, int width)
{
		
		for(int i =0;i<HS_ANGLES;i++)
		{
			for(int j = 0;j<width;j++)
			{
				cout<<array[i*width + j]<<"\t";

			}
			cout<<endl;

		}

}

int getMaximum(unsigned int* const array, int width)
{

	int maximum = *(array + 0);

	for(int i =0;i<HS_ANGLES;i++)
	{
		for(int j =0 ;j<width;j++)
		{
			if(array[i*width + j] > maximum)
				maximum = array[i*width + j];
		

		}
			

	}
	return maximum;

}

/*__global__ void Hough(unsigned char const* const image, unsigned int const
		threshold, unsigned int* const houghspace_1, unsigned int* const houghspace_2)
{
	int const x = blockIdx.x*blockDim.x + threadIdx.x;
	int const y = blockIdx.y*blockDim.y + threadIdx.y;
	__shared__ float sh_m_array[THREADS_X*THREADS_Y];
	int const n = threadIdx.y*THREADS_X + threadIdx.x;

	//Debugging
	//printf("n value : %d \n", n);


	sh_m_array[n]  =  (n-((HS_ANGLES-1)/2.0f)) / (float)((HS_ANGLES-1)/2.0f);
	//printf("shared_array_value : %f \t at postion : %d with thread indexes x: \
	//		%d and \t y : %d \n",sh_m_array[n], n, threadIdx.x, threadIdx.y);
	__syncthreads();

	unsigned char pixel = image[y*IMG_WIDTH + x];
	if(pixel >= threshold)
	{
		for(int n = 0;n<HS_ANGLES;n++)
		{
			float const m = sh_m_array[n];
			int const b1 = x - (int)(y*m) + IMG_HEIGHT;
			int const b2 = y - (int)(x*m) + IMG_WIDTH;
		
			atomicAdd(&houghspace_1[n*HS_1_WIDTH+b1], 1);
			atomicAdd(&houghspace_2[n*HS_2_WIDTH+b2], 1);
		}
	}

	

}
*/

__device__ static int g_counter;
extern __shared__ int shmem[];

__global__ void getNonzeroEdgepoints(unsigned char const* const image, unsigned int* const list)
{

	
	__shared__ unsigned int s_queues[4][32 * PIXELS_PER_THREAD];
	__shared__ int s_qsize[4];
	__shared__ int s_globStart[4];

	const int x = blockIdx.x * blockDim.x * PIXELS_PER_THREAD + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(threadIdx.x == 0)
		s_qsize[threadIdx.y] = 0;
	__syncthreads();

	if(y < 224)
	{	
		const unsigned char* srcRow = image + y*IMG_WIDTH;
		for(int i = 0,xx = x; i<PIXELS_PER_THREAD && xx < 192;++i,xx +=
				blockDim.x)
		{
			if(srcRow[xx])
			{
				const unsigned int val = (y<<16)|xx;
				//Atomic
				const int qidx = atomicAdd(&s_qsize[threadIdx.y],1);
				s_queues[threadIdx.y][qidx] = val;


			}


		}

	}

	__syncthreads();

	if(threadIdx.x == 0 && threadIdx.y == 0 )
	{	
		int totalSize = 0;
		for(int i =0;i<blockDim.y;++i)
			{
				s_globStart[i] = totalSize;
				totalSize += s_qsize[i];	

			}
		
		const int global_Offset = atomicAdd(&g_counter, totalSize);
		for(int i  =0 ;i<blockDim.y;++i)
			s_globStart[i] += global_Offset;
	}

	__syncthreads();

	const int qsize = s_qsize[threadIdx.y];
	int gidx = s_globStart[threadIdx.y] +  threadIdx.x;
	for(int i = threadIdx.x; i<qsize; i+=blockDim.x, gidx +=blockDim.x)
	{
		list[gidx] = s_queues[threadIdx.y][i];

	}

}

__global__ void getLines(unsigned int* const list, const int count, int*
		hough_space,const float irho, const float theta, const int numrho)
{

	int* smem = (int*)shmem;
	for(int i =threadIdx.x; i< numrho + 1;i+=blockDim.x)
		smem[i] = 0;
	__syncthreads();

	const int n = blockIdx.x;
	const float ang = n*theta;
	
	printf("Angle Values : %f \n", ang);
//	printf("Inside Kernel");
	
	/*
	float sinVal;
	float cosVal;

	sincosf(ang, &sinVal, &cosVal);
	sinVal *= irho;
	cosVal *= irho;

	const int shift = (numrho -1)/2;

	for(int i  = threadIdx.x; i<count; i+= blockDim.x)
	{
		const unsigned int val = list[i];
		const int x = (val & 0x0000FFFF);
		const int y = (val>>16) & 0x0000FFFF;


	}
	
	*/

}


/*__global__ void test_kernel(void)
{

	int x = threadIdx.x;
	printf("%d \n", x);


}

*/
void houghTransform(unsigned char const* const edges,const int numangle, const
		int numrho,float thetaStep, float
		rStep)
{
		unsigned char* gimage;	
		unsigned int* glist; 

		void* counterPtr;
		hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter));
		hipError_t c_err;


		c_err = hipMemset(counterPtr,0,sizeof(int));
	
		if(c_err != hipSuccess)
		{
			printf("%s in %s at line %d \n", hipGetErrorString(c_err),
				__FILE__,__LINE__);
			exit(EXIT_FAILURE);
		}


		hipFuncSetCacheConfig(reinterpret_cast<const void*>(getNonzeroEdgepoints), hipFuncCachePreferShared);
			
		c_err = hipMalloc((void**)&gimage, IMG_SIZE*sizeof(unsigned char));
		
		if(c_err != hipSuccess)
		{
			printf("%s in %s at line %d \n", hipGetErrorString(c_err),
				__FILE__,__LINE__);
			exit(EXIT_FAILURE);
		}
		
		c_err = hipMalloc((void**) &glist, IMG_SIZE*sizeof(unsigned int));

		if(c_err != hipSuccess)
		{
			printf("%s in %s at line %d \n", hipGetErrorString(c_err),
				__FILE__,__LINE__);
			exit(EXIT_FAILURE);
		}
		
		/*Copy Image to GPU */	
	
		c_err = hipMemcpy(gimage, edges, IMG_SIZE*sizeof(unsigned char),
			hipMemcpyHostToDevice);

		if(c_err != hipSuccess)
		{
			printf("%s in %s at line %d \n", hipGetErrorString(c_err),
				__FILE__,__LINE__);
			exit(EXIT_FAILURE);
		}

		
		dim3 dimBlock1(THREADS_X, THREADS_Y);
		//dim3 dimGrid1(BLOCKS_X, BLOCKS_Y);
		dim3 dimGrid1(1, 56);
		getNonzeroEdgepoints<<<dimGrid1,dimBlock1>>>(gimage, glist);

		c_err = hipGetLastError();
		if(c_err != hipSuccess)
		{
			printf("Error: %s\n", hipGetErrorString(c_err));
		
		}
		
		hipDeviceSynchronize();

		int totalCount ;
		hipMemcpy(&totalCount, counterPtr, sizeof(int),
				hipMemcpyDeviceToHost);
		cout<<"Total Count :"<<totalCount<<endl;

		if(debug)
		{
			unsigned int* clist = (unsigned int*)malloc(totalCount*sizeof(unsigned int));
			c_err = hipMemcpy(clist, glist, totalCount*sizeof(unsigned int),hipMemcpyDeviceToHost);
			if(c_err != hipSuccess)
			{
					printf("%s in %s at line %d \n", hipGetErrorString(c_err),
				__FILE__,__LINE__);
					exit(EXIT_FAILURE);
			}
			for(int i = 0; i< totalCount; i++)
			{	
				unsigned int const q_value = clist[i];
				cout<<"q_value : "<<q_value<<endl;
				unsigned int const x = (q_value & 0x0000FFFF);
				unsigned int const y = (q_value >> 16) & 0x0000FFFF;
				cout<<"coordinate ("<<x<<","<<y<<")"<<endl;
				cout<<"Value at coordinate :"<<(int)*(edges + y*IMG_WIDTH + x)<<endl;
			}

		
		}

		//Initialize hough_space
		int hough_size = (numangle + 2)*(numrho + 2);	
		int rhspace = numangle + 2;
		int colhspace = numrho + 2;
		int* hough_space = (int*)calloc(hough_size, sizeof(int));
		
		const dim3 block(1024);
		const dim3 grid(rhspace -2);

		//smemSize should be less than 49152 bytes

		size_t smemSize = (colhspace - 1)*sizeof(int);
		cout<<smemSize<<endl;

		thetaStep = thetaStep*(CV_PI/180);
	
		/*Allocate houghSpace on Gpu*/
		int *d_hough_space;

		c_err = hipMalloc((void**)&d_hough_space,hough_size*sizeof(int));
		if(c_err != hipSuccess)
		{
			printf("%s in %s at line %d \n", hipGetErrorString(c_err),
				__FILE__,__LINE__);
			exit(EXIT_FAILURE);
		}
		

		c_err = hipMemset(d_hough_space, 0, hough_size*sizeof(int));
		
		if(c_err != hipSuccess)
		{
			printf("%s in %s at line %d \n", hipGetErrorString(c_err),
				__FILE__,__LINE__);
			exit(EXIT_FAILURE);
		}

		//hipFuncSetCacheConfig(reinterpret_cast<const void*>(getLines),hipFuncCachePreferShared);
		
		getLines<<<grid,block,smemSize>>>(glist, totalCount,d_hough_space, 1.0f/
				rStep, thetaStep, colhspace -2);
		
		c_err = hipGetLastError();	
		if(c_err != hipSuccess)
		{
			printf("Error: %s\n", hipGetErrorString(c_err));
		
		}	

		hipDeviceSynchronize();

		


}












int main(int argc, char* argv[])
{

	Mat src_host = imread("/home/nvidia/Binary_test_image_for_cuda_ht.png",
			CV_8UC1);
	cout<<"cols"<<src_host.cols<<endl;
	cout<<"rows"<<src_host.rows<<endl;

	//cout<<src_host<<endl;
	//cout<<src_host.at<unsigned int>(48,34)<<endl;
	int count = 0;
	//cout<<src_host<<endl;
		
	count = countNonZero(src_host);
	cout<<count<<endl;

	Size size = src_host.size();
	int width = size.width;
	int height = size.height;

	if(debug)
	{
		imshow("Result",src_host);
		waitKey(0);
		Size size = src_host.size();
		cout<<size<<endl;
		int width = size.width;
		int height = size.height;	
		cout<<width<<endl;
		cout<<height<<endl;	
	}

	/*Convert array to uchar* (0-255)*/	
	unsigned char *edge_image = src_host.data;
	if(debug)
	{
		print_image(edge_image, height,width);	
	
	}
	//unsigned char* rowptr = edge_image + 2*IMG_WIDTH;
	//cout<<(int)*rowptr<<endl;

	/*unsigned int* houghspace_gpu_1 = (unsigned int*)malloc(HS_1_SIZE*sizeof(unsigned int));
	unsigned int* houghspace_gpu_2 = (unsigned int*)malloc(HS_2_SIZE*sizeof(unsigned int));
	
	unsigned int const threshold = 50;

	houghTransform(edge_image, threshold, houghspace_gpu_1, houghspace_gpu_2);	
	*/
		
	float rMin = 0;
	float rMax = (IMG_WIDTH + IMG_HEIGHT)*2 + 1;
	float rStep = 1.0;

	float thetaMin = 0;
	float thetaMax = 180;
	float thetaStep = 1;
	
	const int numangle = std::round((thetaMax - thetaMin)/thetaStep);
	const int numrho = std::round(rMax/rStep);

	if(1)
	{
		cout<<numangle<<endl;
		cout<<numrho<<endl;
	}

	float* r_values = new float[numrho];
	float* th_vaues = new float[numangle];
	
	int ri, thetai;
	float r, theta;

	for(r = rMin + rStep/2, ri=0;ri<numrho;ri++,r +=rStep)
	{
		r_values[ri] = r;

	}

	for(theta = thetaMin, thetai =0;thetai<numangle;thetai++,theta
			+=thetaStep)
	{
		th_vaues[thetai] =theta;

	}

	if(debug)
	{
		print_array(r_values, numrho);
		print_array(th_vaues, numangle);
	}
	
	//int count = countNonZero(src_host);
	//cout<<count<<endl;	
	
	houghTransform(edge_image, numangle, numrho,thetaStep, rStep);
	
	

	







}
