#include "hip/hip_runtime.h"
#include"bev_thrust.hpp"

bool debug = false;


/*Functor For Stream Compaction*/

/*
struct checkSanity
{
	__host__ __device__
		bool operator()(tuple_t x)
		{
			return(thrust::get<0>(x) >= 1 && thrust::get<1>(x) >=1 &&
					thrust::get<0>(x) <= IMAGE_WIDTH && thrust::get<1>(x) <= IMAGE_HEIGHT);
		}

};
*/

struct checkSanity
{
	__host__ __device__
		bool operator()(const float x)
		{
			return (x >=  1 && x <= IMAGE_WIDTH);
		}


};

void print1dvector(row_t vector)
{   
	for(const auto& elem: vector)
		cout<<setprecision(11)<<elem<<"\t";
	
	cout<<endl;

}

void print2dvector(matrix_t vector)
{   
	for(const auto& row : vector)
	{
		for(const auto& elem : row)
		{
			cout<<setprecision(11)<<elem<<"\t";
							
		}
		cout<<endl;
	}
	cout<<endl;
}



matrix_t  matrix_multiplication(matrix_t const& vec_a, matrix_t  const& vec_b)
{
	int vec_a_rows = vec_a.size();
	int vec_a_columns = vec_a[0].size();
	
	int vec_b_rows = vec_b.size();
	int vec_b_columns  = vec_b[0].size();
			
	if(debug)
	{ 
		cout<<vec_a_rows<<"\t"<<vec_a_columns<<endl;
		cout<<vec_b_rows<<"\t"<<vec_b_columns<<endl;
	}
				
	matrix_t R2_1(vec_a_rows, row_t(vec_b_columns,0));
	
	for(int i =0;i<vec_a_rows;++i)
	{   
		for(int j =0;j<vec_b_columns;++j)
		{
			R2_1[i][j] = 0;
			for(int k = 0;k<vec_a_columns;++k)
			{
				R2_1[i][j] = R2_1[i][j] + (vec_a[i][k]*vec_b[k][j]);
				
			}
		}			
	}
	return R2_1;

}

void getCofactor(matrix_t &vec_a, matrix_t &vec_b, int p,int q, int vec_a_rows)
{
	int i = 0, j = 0;
	for( int row = 0; row < vec_a_rows; row++)
	{
		for( int col = 0; col < vec_a_rows ; col++)	
		{
			if( row != p && col !=q)
			{
				vec_b[i][j++] = vec_a[row][col];
				if(j == vec_a_rows -1)
				{
					j = 0;
					i++;
				}
			}	
		}
	}
}


double determinant(matrix_t &vec_a, int n)
{   
	double D = 0.0;
	if(n==1)
		return (double) vec_a[0][0];
				
	matrix_t temp(4, row_t(4));
	//print2dvector(temp);
			
	int sign = 1;
	for( int f = 0;f< n;f++)
	{
		getCofactor(vec_a, temp,0,f,n);
		D += sign*(vec_a[0][f])*(determinant(temp,n-1));
		sign = -sign;
	}
	
	return D;
}

matrix_t adjoint(matrix_t &vec_a)
{
	int vec_a_rows = vec_a.size();
	int vec_a_columns = vec_a[0].size();
	
	int sign = 1;
	matrix_t temp(vec_a_rows, row_t(vec_a_columns,0));
	matrix_t adj(vec_a_rows, row_t(vec_a_rows,0));
		
	for(int i = 0; i<vec_a_rows;i++)
	{
		for(int j = 0;j<vec_a_columns;j++)
		{
			getCofactor(vec_a, temp, i, j, vec_a_rows);
			sign  = ((i+j)%2==0)? 1: -1;
			adj[j][i] = (sign)*(determinant(temp, vec_a_rows -1));
		}
	}
	return adj;
			
}


matrix_t inverse(matrix_t &vec_a)
{
	int vec_a_rows = vec_a.size();
 	
	double det = determinant(vec_a,vec_a_rows);
	
	matrix_t adj(vec_a_rows, row_t(vec_a_rows,0));
	matrix_t inverse_matrix(vec_a_rows, row_t(vec_a_rows,0));
			
	adj = adjoint(vec_a);
	
	for(int i = 0;i<vec_a_rows;i++)
	{
		for( int j = 0;j<vec_a_rows;j++)
		{
			inverse_matrix[i][j] = adj[i][j]/det;
		}
	}
	return inverse_matrix;


}


__global__ void matrix_mul(float* d_A, float* d_B, float* d_C, int numARows, int numAColumns, int numBRows, int numBColumns,
		int numCRows, int numCColumns)
{
	__shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
 	__shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];	

	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int Row = by*TILE_WIDTH + ty;
	int Col = bx*TILE_WIDTH + tx;

	float Pvalue = 0;

	#pragma unroll
	for (int m = 0; m < (numAColumns-1)/TILE_WIDTH+1; ++m)
	{
		if (Row < numARows && m*TILE_WIDTH+tx < numAColumns)
		{
			ds_M[ty][tx] = d_A[Row*numAColumns + m*TILE_WIDTH+tx];
	
		}
		else
		{
			ds_M[ty][tx] = 0;

		}

		if (Col < numBColumns && m*TILE_WIDTH+ty < numBRows)
		{
			ds_N[ty][tx] = d_B[(m*TILE_WIDTH+ty)*numBColumns+Col];
		}
		else
		{
			ds_N[ty][tx] = 0;
		}

		__syncthreads();

		for (int k = 0; k < TILE_WIDTH; ++k)
		{
			Pvalue += ds_M[ty][k] * ds_N[k][tx];
		}

		__syncthreads();
	
	}

	if (Row < numCRows && Col < numCColumns)
		d_C[Row*numCColumns+Col] = Pvalue;

}


float* getMatrix(matrix_t Tr33, float* h_B, int numBRows, int numBColumns)
{
	double e1 = getTickCount();

	float* h_A;
	float* h_C; 
	float* d_A;
	float* d_B;
	float* d_C;

	int numARows = Tr33.size();
	int numAColumns = Tr33[0].size();

	int numCRows = numARows;
	int numCColumns = numBColumns;

	h_A = (float*)malloc(numARows*numAColumns*sizeof(float));

	/*Populate h_A and h_B */
	for(int i = 0 ;i<numARows;i++)
	{	
		for(int j =0;j<numAColumns;j++)
		{
			*(h_A + i*numAColumns + j) = Tr33[i][j];

		}

	}
	
	h_C = (float*)malloc(numCRows*numCColumns*sizeof(float));

	double e3 = getTickCount();
	//hipSetDevice(0);
	hipMalloc((void**)&d_A, sizeof(float) * numARows * numAColumns);
	hipMalloc((void**)&d_B, sizeof(float) * numBRows * numBColumns);
	hipMalloc((void**)&d_C, sizeof(float) * numCRows * numCColumns);
	double e4 = getTickCount();
	double time_alloc = (e4 - e3)/getTickFrequency();

	cout<<"Time for mem alloc"<<time_alloc<<endl;

	hipMemcpy(d_A, h_A, sizeof(float) * numARows * numAColumns, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeof(float) * numBRows * numBColumns, hipMemcpyHostToDevice);

	int dim_grid_x = (numCColumns - 1)/TILE_WIDTH +1;
	int dim_grid_y = (numCRows -1)/TILE_WIDTH + 1;

	//double e2 = getTickCount();
	//double time = (e2 -e1)/getTickFrequency();
	//cout<<"Time for allocation"<<time<<endl;

	dim3 dimGrid(dim_grid_x, dim_grid_y);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

	matrix_mul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
	
	hipMemcpy(h_C, d_C, sizeof(float) * numCRows * numCColumns, hipMemcpyDeviceToHost);

	//hipDeviceSynchronize();

	if(debug)
	{
		for(int i = 0 ;i<numCRows;i++)
		{
			for(int j  = 0;j<numCColumns;j++)
			{
				cout<<*(h_C + i*numCRows + j)<<"\t";
			}
		cout<<endl;
		}
	}

	//double e2 = getTickCount();
	//double time = (e2 -e1)/getTickFrequency();
	
	//cout<<"Time for allocation"<<time<<endl;


	float* row_ptr_0 = h_C;
	float* row_ptr_1 = h_C + numCColumns;
	float* row_ptr_2 = h_C + 2*numCColumns;

	for(int i =0;i<numCColumns;i++)
	{
		*(row_ptr_0 +  i) =  *(row_ptr_0 + i)/(*(row_ptr_2 + i));
		*(row_ptr_1 + i) = *(row_ptr_1 + i)/(*(row_ptr_2 + i));
		*(row_ptr_2 + i) = *(row_ptr_2 + i)/(*(row_ptr_2 + i));

	}


	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	double e2 = getTickCount();
	double time = (e2 -e1)/getTickFrequency();
	
	cout<<"Time for allocation"<<time<<endl;
	
	return h_C;



}

BevParams::BevParams(float bev_res, tuple_int bev_xLimits, tuple_int bev_zLimits, tuple_int imSize)
{
	
	//this->bev_size = make_tuple(round((get<1>(bev_zLimits) - get<0>(bev_zLimits))/bev_res),round((get<1>(bev_xLimits) -get<0>(bev_xLimits))/bev_res));
	this->bev_size = {static_cast<int>(std::round((bev_zLimits.b - bev_zLimits.a)/bev_res)), static_cast<int>(std::round((bev_xLimits.b - bev_xLimits.a)/bev_res))};	
	this->bev_res = bev_res;
	this->bev_xLimits = bev_xLimits;
	this->bev_zLimits = bev_zLimits;
	this->imSize = imSize;

};






Calibration::Calibration()
{
	


};


void Calibration::setup_calib(matrix_t P2, matrix_t R0_rect, matrix_t Tr_cam_to_road)
{
	double e1 = getTickCount();
	this->P2 = P2;
	(this->R0_Rect).resize(4,row_t(4,0));

	
	/*
	* Slower than iterator
	for(int i = 0;i<R0_rect.size();i++)
	{
		std::copy(R0_rect[i].begin(), R0_rect[i].end(), (this->R0_Rect)[i].begin());
	}
	*/
	
	matrix_t::iterator row,i;
	row_t::iterator column,j;
	
	for(row = (this->R0_Rect).begin(), i = R0_rect.begin() ; row != (this->R0_Rect).end() - 1;++row,++i)
	{
		for(column = row->begin(), j = i->begin(); column != row->end()-1;++column,++j)
		{
			*column = *j;	
		}
	}
			
	(this->R0_Rect)[3][3] = 1.0;
	
	matrix_t R2_1 = matrix_multiplication(this->P2, this->R0_Rect);
	
	Tr_cam_to_road.resize(4, row_t(4,0));
	Tr_cam_to_road[3][3] = 1;
	this->Tr_cam_to_road = Tr_cam_to_road;

	matrix_t Tr_cam_to_road_inverse = inverse(this->Tr_cam_to_road);
	this->Tr = matrix_multiplication(R2_1, Tr_cam_to_road_inverse);
	
	unsigned columntoDelete = 1;

	for(unsigned i = 0;i<(this->Tr).size();++i)
	{
		if((this->Tr)[i].size() > columntoDelete)
		{
			(this->Tr)[i].erase((this->Tr)[i].begin() + columntoDelete);
		}


	}
	this->Tr33 = this->Tr;
	
	double e2 = getTickCount();
	double time =  (e2 - e1)/getTickFrequency();

	cout<<"Time for Calibration"<<time<<endl;

}



matrix_t Calibration::get_matrix33()
{   
	return this->Tr33;

}









void  BirdsEyeView::setup(matrix_t P2, matrix_t R0_rect, matrix_t Tr_cam_to_road)
{
	(this->calib)->setup_calib(P2, R0_rect, Tr_cam_to_road);
	this->set_matrix33((this->calib)->get_matrix33());

}

void BirdsEyeView::set_matrix33(matrix_t Tr33)
{   
	this->Tr33 = Tr33;
}

void BirdsEyeView::initialize(Mat& image)
{
	double e6 =  getTickCount();

	Size size =  image.size();
	this->imSize = {size.width, size.height};
	
	float res = (this->bevParams)->bev_res;

	int x_vec_length = ((this->bevParams)->bev_xLimits.b - ((this->bevParams)->bev_xLimits.a + res/2))/res + 1;
	int z_vec_length = ((this->bevParams)->bev_zLimits.b - res/2 - (this->bevParams)->bev_zLimits.a)/res + 1;
		
	double init_value_x = (this->bevParams)->bev_xLimits.a + res/2;
	double init_value_z = (this->bevParams)->bev_zLimits.b - res/2;
	
	double e1 = getTickCount();
	
	row_t x_vec(x_vec_length), z_vec(z_vec_length);
	
	for(int i = 0 ;i<x_vec_length;i++)
	{
		x_vec[i] = init_value_x;
		init_value_x += res;
	}
	
	for(int i = 0 ;i<z_vec_length;i++)
	{
		z_vec[i] = init_value_z;
		init_value_z -= res;
	}

	double e2 = getTickCount();

	double time = (e2 - e1)/getTickFrequency();
	cout<<"Time to fill vectors"<<time<<endl;

	/*
	h_row_t x_vec(x_vec_length), z_vec(z_vec_length);
	
	for(int i =0 ;i<x_vec_length;i++)
	{
		x_vec[i] = init_value_x;
		init_value_x +=res;
			
	}
	for(int i =0;i<z_vec_length;i++)
	{

		z_vec[i] = init_value_z;
		init_value_z -=res;
	}
	double e2 = getTickCount();

	double time = (e2 - e1)/getTickFrequency();
	cout<<"Time to fill vectors"<<time<<endl;
	*/

	
	int vec_size = (this->bevParams)->bev_size.a*(this->bevParams)->bev_size.b;
	row_t z_mesh_vec(vec_size), x_mesh_vec;
	
	for(int i = 0;i<vec_size;i++)
	{
		int temp = i%z_vec_length;
		z_mesh_vec[i] = z_vec[temp];
	}
		int i = 0;
	while(i < x_vec_length)
	{
		for(int j = 0;j< z_vec_length ;j++)
		{
			x_mesh_vec.push_back(x_vec[i]);
			
		}
		i++;
	}
	
	row_t y_world(vec_size, 1.0);
	matrix_t uv_mat(3, row_t(vec_size, 0));

	row_t::iterator row_x, row_z, row_y, row_uv_0, row_uv_1,row_uv_2;
	
	double e3 = getTickCount();
	
	/*Highly Optimized than std::copy*/
	for(row_x = x_mesh_vec.begin(), row_z = z_mesh_vec.begin(), 
		row_y  = y_world.begin(), row_uv_0 = uv_mat[0].begin(), row_uv_1 = uv_mat[1].begin(),
		row_uv_2 = uv_mat[2].begin(); row_x != x_mesh_vec.end(); row_x++, row_z++, row_y++, row_uv_0++, row_uv_1++, row_uv_2++)
	{
		*row_uv_0 = *row_x;
		*row_uv_1 = *row_z;
		*row_uv_2 = *row_y;
	}
	

	double e4 = getTickCount();
	
	double time_to_populate = (e4 -e3)/getTickCount();
	cout<<"Time to Populate \t"<<time_to_populate<<endl;

	this->uvMat = uv_mat;

	double e7 = getTickCount();
	double time_for_init = (e7-e6)/getTickFrequency();

	cout<<"Time for init"<<time_for_init<<endl;
	
	this->numBRows = uv_mat.size();
	this->numBColumns = uv_mat[0].size();

	this->h_B = (float*)malloc(numBRows*numBColumns*sizeof(float));

	for(int i = 0;i<numBRows;i++)
	{
		for(int j =0;j<numBColumns;j++)
		{
			*(this->h_B + i*numBColumns + j) = uvMat[i][j]; 

		}
	}


	vector<int> values_z((this->bevParams)->bev_size.a);
	iota(values_z.begin(), values_z.end(), 1);

	for(int i = 0; i< x_vec_length ;i++)
	{
		(this->z_index_vec).insert(z_index_vec.end(), values_z.begin(),
		values_z.end());
	
	}

	vector<int> values_x((this->bevParams)->bev_size.b);
	iota(values_x.begin(), values_x.end(),1 );

	int index = 0;
		
	while(index < x_vec_length)
	{
		for(int i =0; i<z_vec_length ;i++)
			(this->x_index_vec).push_back(values_x[index]);
		index++;
	}

	
	if(debug)
	{
		for(const auto& i : x_index_vec)
			cout<<i<<"\t";
    	cout<<endl;
	}

	/*dummy hipMalloc to init device*/
	float* d_B;
	hipMalloc((void**)&d_B, sizeof(float)*this->numBRows*this->numBColumns);
	hipFree(d_B);




}





BirdsEyeView::BirdsEyeView(float bev_res, double invalid_value, tuple_int bev_xRange_minMax, tuple_int bev_zRange_minMax)
{			
		
	this->calib = new Calibration();
	this->bev_res = bev_res;
	this->invalid_value = invalid_value;
	this->bev_xRange_minMax = bev_xRange_minMax;
	this->bev_zRange_minMax = bev_zRange_minMax;
	this->bevParams = new BevParams(bev_res, bev_xRange_minMax, bev_zRange_minMax, this->imSize);
		
}

void BirdsEyeView::computeLookUpTable(Mat& image)
{

	double e1 = getTickCount();
	float* result = getMatrix(this->Tr33, this->h_B,this->numBRows, this->numBColumns);
	

	if(debug)
	{
		for(int i =0 ;i<3;i++)
		{
			for(int j  = 0;j<80000;j++)
			{
				cout<<*(result  + i*80000 + j)<<"\t";
			}
			
			cout<<endl;
		}
	}
	
	int numCol = this->numBColumns;

	row_t xi_1(numCol), yi_1(numCol);

	float* result_row_0 = result;
	float* result_row_1 = result + 1*numCol;
	
	vector<int> x_index_vec_copy = this->x_index_vec;
	vector<int> z_index_vec_copy = this->z_index_vec;
	
	vector<int> z_vec_sel(numCol), x_vec_sel(numCol);

	/*
	double e3 = getTickCount();
	
	device_vector<float> result_row_0(result, result + numCol);
	device_vector<float> xi_1(numCol);

	auto it = thrust::copy_if(result_row_0.begin(),result_row_0.end(),xi_1.begin(), checkSanity());
			
	host_vector<float> h_xi_1 (xi_1.begin(),it);
	/

	//double e4 = getTickCount();
	//double time = (e4-e3)/getTickFrequency();
//	cout<<"Time for comapction"<<time<<endl;



	device_vector<float> result_row_0(result, result + numCol);
	device_vector<float> result_row_1(result + numCol,result + numCol);

	zipIterator result_begin(thrust::make_tuple(result_row_0.begin(),
				result_row_1.begin()));
	zipIterator result_end(thrust::make_tuple(result_row_0.end(),
				result_row_1.end()));
	device_vector<float> xi_1;
	device_vector<float> yi_1;
	
	zipIterator uv_begin(thrust::make_tuple(xi_1.begin(),yi_1.begin()));
	
	thrust::copy_if(result_begin, result_end, uv_begin, checkSanity());

*/
	int count  = 0;

	for(int i =0;i<numCol;i++)
	{
		if((*(result_row_1 + i) >=1) & (*(result_row_0 +i) >=1) &
						(*(result_row_1 + i) <= this->imSize.b) &
						(*(result_row_0 + i) <= this->imSize.a))
		{
			xi_1[count] = *(result_row_0 + i);
			yi_1[count] = *(result_row_1 + i);
			z_vec_sel[count] = z_index_vec_copy[i];
			x_vec_sel[count] = x_index_vec_copy[i];
			count++;	
		}

	}


	z_vec_sel.resize(count);
	x_vec_sel.resize(count);
	xi_1.resize(count);
	yi_1.resize(count);
	

	if(debug)
	{
		for(const auto& i: xi_1)
			cout<<i<<"\t";
		cout<<endl;
	}
	
	if(debug)
	{
		for(const auto& i : z_vec_sel)
			cout<<i<<"\t";
    	cout<<endl;
	}
	
	
	
	vector<int>::const_iterator m,k;
	row_t::const_iterator i,j;

	Mat output_image((this->bevParams)->bev_size.a,(this->bevParams)->bev_size.b,CV_8UC1);

	unsigned char* i_im = image.data;
	unsigned char* o_im = output_image.data;


	for(i = xi_1.begin() , j = yi_1.begin(), m = x_vec_sel.begin(), k = z_vec_sel.begin();i != xi_1.end();i++,j++,m++,k++)
	{
		int row = (int)*j -1;
		int column = (int)*i -1;

		int row_output_image = (int)*k -1;
		int column_output_image = (int)*m -1;
	
		//cout<<"Row \t"<<row<<"Column \t"<<column<<endl;

		*(o_im + row_output_image*200 + column_output_image) = *(i_im + row*1242+ column);

	}

	imshow("result", output_image);
	waitKey(0);

	double e2 = getTickCount();
	double time_lookup = (e2 -e1)/getTickFrequency();
	cout<<"Time for lookup \t"<<time_lookup<<endl;
}










int main(int argc, char* argv[])
{		
	Mat test_image = imread("/home/nvidia/Lane_Detection/Original_Images/img_0.png", CV_LOAD_IMAGE_GRAYSCALE);
	
	if(debug)
	{
		imshow("Test_image", test_image);
		waitKey(0);
	
	}
	/*define Parameters*/
	float bev_res = 0.1;
	tuple_int bev_xRange_minMax = {-10,10};
	tuple_int bev_zRange_minMax = {6, 46};
	float invalid_value = -numeric_limits<float>::infinity(); 

	BirdsEyeView bev(bev_res, invalid_value,bev_xRange_minMax, bev_zRange_minMax);
	

/*Projection matrix for left color camera in rectified coordinates*/
/*3x4*/
	matrix_t P2
	{
		{7.215377000000e+02, 0.000000000000e+00, 6.095593000000e+02, 4.485728000000e+01},
		{0.000000000000e+00 ,7.215377000000e+02 ,1.728540000000e+02 ,2.163791000000e-01},
		{0.000000000000e+00, 0.000000000000e+00, 1.000000000000e+00, 2.745884000000e-03}
		
	};
	
	/*Initialize Rotation Matrix (3x3) */
	matrix_t R0_rect
	{
		{9.999239000000e-01, 9.837760000000e-03, -7.445048000000e-03},
		{-9.869795000000e-03, 9.999421000000e-01,-4.278459000000e-03},
		{7.402527000000e-03, 4.351614000000e-03, 9.999631000000e-01}
	};

	/*Rigid transformation from (non-rectified) camera to road coordinates (3x4)*/
	matrix_t Tr_cam_to_road
	{
		{9.999044710077e-01,-1.170165577363e-02, -7.360826724365e-03, 1.911984983337e-02},
		{1.160251828357e-02,9.998432738993e-01,-1.336987011872e-02,-1.562198078590e+00},
		{7.516122576373e-03, 1.328318612284e-02, 9.998834806284e-01,2.752775890648e-01}
	
	};


	bev.setup(P2, R0_rect, Tr_cam_to_road);
	
	bev.initialize(test_image);
	bev.computeLookUpTable(test_image);


}
